#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#include "berkelib_cuda.h"
#include "reducer.cuh"

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS


inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

template <typename scalar_t>
__global__ void max_mul_kernel(
    const scalar_t* __restrict__ src,
    const int64_t* __restrict__ edge_start, 
    const int64_t* __restrict__ edge_end,
    scalar_t* __restrict__ res,
    size_t hidden_dim,
    size_t N)
{
    
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(thread_id < N){
        int edge_index = thread_id / hidden_dim;
        int hidden_dim_index = thread_id % hidden_dim;      

        Reducer<scalar_t, MAX>::atomic_write(
           res + edge_end[edge_index]*hidden_dim + hidden_dim_index, 
            src[ edge_start[edge_index]*hidden_dim + hidden_dim_index]);  
    }
}

template <typename scalar_t>
__global__ void max_mul_arg_kernel(
    const scalar_t* __restrict__ src,
    const int64_t* __restrict__ edge_start, 
    const int64_t* __restrict__ edge_end,
    scalar_t* __restrict__ res,
    int64_t* __restrict__ arg_out,
    size_t hidden_dim,
    size_t N)
{
    
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(thread_id < N){
        int edge_index = thread_id / hidden_dim;
        int hidden_dim_index = thread_id % hidden_dim;

        if(res[edge_end[edge_index]*hidden_dim + hidden_dim_index] == src[edge_start[edge_index]*hidden_dim + hidden_dim_index]){
            arg_out[edge_end[edge_index]*hidden_dim + hidden_dim_index] = edge_start[edge_index];
        }
    }
}

std::tuple<torch::Tensor,torch::Tensor> max_mul_cuda_forward(
    const torch::Tensor src, 
    const torch::Tensor edge_start, 
    const torch::Tensor edge_end,
    int64_t res_dim)
{
    //check input
    CHECK_INPUT(src);
    CHECK_INPUT_DIM(edge_start.size(0) == edge_end.size(0));
    CHECK_INPUT(edge_start);
    CHECK_INPUT(edge_end);

    size_t hidden_dim = size(src, 1);
    size_t N = edge_end.numel()*hidden_dim;

    //create out and arg_out Tensor with given out_dim
    auto res_dims = src.sizes().vec();
    res_dims[0] = res_dim;
    torch::Tensor res = torch::empty(res_dims, src.options());
    torch::Tensor arg_out = torch::full_like(res,src.size(0),edge_start.options());
    
    AT_DISPATCH_FLOATING_TYPES(src.type(), "_", [&] {
        res.fill_(std::numeric_limits<scalar_t>::lowest());
        auto src_data = src.data_ptr<scalar_t>();
        auto res_data = res.data_ptr<scalar_t>();
        auto arg_out_data = arg_out.data_ptr<int64_t>();
        auto edge_start_data = edge_start.data_ptr<int64_t>();
        auto edge_end_data = edge_end.data_ptr<int64_t>();

        max_mul_kernel<scalar_t><<<BLOCKS(N), THREADS>>>(
            src_data,
            edge_start_data,
            edge_end_data,
            res_data,
            hidden_dim,
            N);

        res.masked_fill_(res == std::numeric_limits<scalar_t>::lowest(), (scalar_t)0);

        max_mul_arg_kernel<scalar_t><<<BLOCKS(N), THREADS>>>(
            src_data,
            edge_start_data,
            edge_end_data,
            res_data,
            arg_out_data,
            hidden_dim,
            N);   
    });

    checkCuda(hipGetLastError());
    
    return std::make_tuple(res,arg_out);   
}